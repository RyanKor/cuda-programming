#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void helloCUDA(void)
{
    printf("Hello CUDA from GPU!\n");
}

int main(void)
{
    printf("Hello GPU from CPU!\n");

    // GPU 커널 호출
    helloCUDA<<<1, 10>>>();

    // GPU 작업 동기화
    hipDeviceSynchronize();

    // CUDA 커널 실행 에러 확인
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA kernel launch error: %s\n", hipGetErrorString(err));
    }

    return 0;
}
